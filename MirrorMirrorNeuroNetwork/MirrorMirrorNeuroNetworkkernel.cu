#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <cstdarg>
#include <vector>
#include <opencv2/opencv.hpp>
#include <Windows.h>

#define RANDBASE1 (long long)0x8000
#define RANDBASE2 (long long)0x40000000
#define RANDBASE3 (long long)0x200000000000

using namespace std;
using namespace cv;

typedef double CalFloat;

CalFloat Sigmoid(CalFloat Val)
{
	if (Val >= 3.00) return 1.00 - 1.00 / (Val * Val * Val);
	if (Val <= -3.00) return -1.00 / (Val * Val * Val);
	return 1.00 / (1.00 + exp(-Val));
}

CalFloat SigmoidGrad(CalFloat Val)
{
	if (Val >= 3.00 || Val <= -3.00) return 3.00 / (Val * Val * Val * Val);
	return Sigmoid(Val) * (1.00 - Sigmoid(Val));
}

double log(double Base, double Antilog)
{
	return log(Antilog) / log(Base);
}

//范围内int随机数
int RandomInt(int minValue, int maxValue)//
{
	if (maxValue - minValue <= 1)
		return minValue;
	return ((rand() + rand() * (RAND_MAX + 1)) % (maxValue - minValue + 1)) + minValue;
}

//范围内long long随机数
long long RandomLongLong(long long minValue, long long maxValue)//
{
	if (maxValue - minValue <= 1)
		return minValue;
	return (((long long)rand() + (long long)rand() * RANDBASE1 + (long long)rand() * RANDBASE2 + (long long)rand() * RANDBASE3) % (maxValue - minValue + 1)) + minValue;
}

//范围内double随机数
double RandomDouble(double minValue, double maxValue)//整数位在1e7范围中，小数精度到7位
{
	const double numscale = 1e7;//先缩放，后还原
	return RandomLongLong((long long)(minValue * numscale), (long long)(maxValue * numscale)) / numscale;
}

//范围内float随机数
float RandomFloat(float minValue, float maxValue)//整数位在1e7范围中，小数精度到7位
{
	const float numscale = 1e3;//先缩放，后还原
	return RandomLongLong((long long)(minValue * numscale), (long long)(maxValue * numscale)) / numscale;
}

struct NeuroNet
{
	int Deep, * Size;
	CalFloat*** w, ** b;
	CalFloat** CalVal;
	NeuroNet()
	{
		Deep = 0;
		Size = NULL;
		w = NULL;
		b = NULL;
	}
	//InitMode == 0 => all 0 init; InitMode == 1 => Random Init
	NeuroNet(vector<int> Scale, CalFloat InitMode = 1)
	{
		Size = (int*)malloc(Scale.size() * sizeof(int));
		for (int i = 0; i < Scale.size(); i++)
			Size[i] = Scale[i];
		Deep = Scale.size() - 1;
		b = (CalFloat**)malloc(Deep * sizeof(CalFloat*));
		for (int i = 0; i < Deep; i++)
		{
			b[i] = (CalFloat*)malloc(Scale[i + 1] * sizeof(CalFloat));
			//memset(b[i], 0, sizeof(Scale[i + 1] * sizeof(CalFloat)));
			for (int j = 0; j < Size[i + 1]; j++)
			{
				b[i][j] = RandomFloat(-1, 1) * InitMode;
			}
		}
		w = (CalFloat***)malloc(Deep * sizeof(CalFloat**));//
		for (int i = 0; i < Deep; i++)
		{
			w[i] = (CalFloat**)malloc(Scale[i] * sizeof(CalFloat*));
			for (int j = 0; j < Scale[i]; j++)
			{
				w[i][j] = (CalFloat*)malloc(Scale[i + 1] * sizeof(CalFloat));
				for (int k = 0; k < Size[i + 1]; k++)
				{
					w[i][j][k] = RandomFloat(-1, 1) * InitMode;
				}
				//memset(w[i][j], 0, sizeof(Scale[i + 1] * sizeof(CalFloat)));
			}
		}
		CalVal = (CalFloat**)malloc(Deep * sizeof(CalFloat*));
		for (int i = 0; i < Deep; i++)
		{
			CalVal[i] = (CalFloat*)malloc(Size[i + 1] * sizeof(CalFloat));
			for (int j = 0; j < Size[i + 1]; j++)
			{
				CalVal[i][j] = 0;
			}
		}
	}
	~NeuroNet()
	{
		free(w);
		free(b);
		free(Size);
		free(CalVal);
	}
	int Run(CalFloat* Input, bool DebugTag = false)
	{
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i + 1]; j++)
			{
				CalFloat TmpVal = 0;
				for (int k = 0; k < Size[i]; k++)
				{
					if (i == 0)
					{
						TmpVal += Input[k] * w[i][k][j];
					}
					else
					{
						TmpVal += CalVal[i - 1][k] * w[i][k][j];
					}
				}
				TmpVal += b[i][j];
				CalVal[i][j] = Sigmoid(TmpVal);
				if (DebugTag) cout << CalVal[i][j] << " ";
			}
			if (DebugTag) cout << "\n";
		}
		CalFloat MaxVal = 0, MaxLabel = 0;//-1e9?
		for (int i = 0; i < Size[Deep]; i++)
		{
			if (MaxVal < CalVal[Deep - 1][i])
			{
				MaxLabel = i;
				MaxVal = CalVal[Deep - 1][i];
			}
		}
		return MaxLabel;
	}
	void BPTrain(CalFloat** Input, int* Output, int OffsetIdx = 0, int n = 1, CalFloat StepK = 0.0010, bool DebugTag = false, string FileName = "BPDebug.txt")
	{
		vector<int> VecSize(0);
		for (int i = 0; i < Deep + 1; i++)
		{
			VecSize.push_back(Size[i]);
		}
		NeuroNet DeltaNet(VecSize, 0);
		for (int ImgIdx = 0; ImgIdx < n; ImgIdx++)
		{
			for (int i = 0; i < Deep; i++)
			{
				for (int j = 0; j < Size[i + 1]; j++)
				{
					CalVal[i][j] = 0;
				}
			}
			for (int i = 0; i < Deep; i++)
			{
				for (int j = 0; j < Size[i + 1]; j++)
				{
					CalFloat TmpVal = 0;
					for (int k = 0; k < Size[i]; k++)
					{
						if (i == 0)
						{
							TmpVal += Input[ImgIdx + OffsetIdx][k] * w[i][k][j];
						}
						else
						{
							TmpVal += CalVal[i - 1][k] * w[i][k][j];
						}
					}
					TmpVal += b[i][j];
					CalVal[i][j] = Sigmoid(TmpVal);
				}
			}
			CalFloat MaxVal = 0, MaxLabel = 0;//-1e9?
			for (int i = 0; i < Size[Deep]; i++)
			{
				if (MaxVal < CalVal[Deep - 1][i])
				{
					MaxLabel = i;
					MaxVal = CalVal[Deep - 1][i];
				}
			}
			for (int i = Deep - 1; i >= 0; i--)
			{
				if (i == Deep - 1)
				{
					for (int j = 0; j < Size[Deep]; j++)
					{
						DeltaNet.CalVal[Deep - 1][j] = (j == Output[ImgIdx + OffsetIdx] ? 1.00 : 0.00);//
					}
				}
				else
				{
					for (int k = 0; k < Size[i + 1]; k++)
					{
						CalFloat CalGrad = 0;
						for (int j = 0; j < Size[i + 1 + 1]; j++)
						{
							CalGrad += 2.00 * (CalVal[i + 1][j] - DeltaNet.CalVal[i + 1][j]) * CalVal[i + 1][j] * w[i + 1][k][j];
						}
						DeltaNet.CalVal[i][k] = CalVal[i][k] - StepK * CalGrad;//+= =
					}
				}//
				for (int j = 0; j < Size[i + 1]; j++)
				{
					DeltaNet.b[i][j] += -2.00 * (CalVal[i][j] - DeltaNet.CalVal[i][j]) * CalVal[i][j] * StepK;
				}
				for (int k = 0; k < Size[i]; k++)
				{
					for (int j = 0; j < Size[i + 1]; j++)
					{
						if (i - 1 < 0) DeltaNet.w[i][k][j] += -2.00 * (CalVal[i][j] - DeltaNet.CalVal[i][j]) * CalVal[i][j] * Input[ImgIdx + OffsetIdx][k] * StepK;
						else DeltaNet.w[i][k][j] += -2.00 * (CalVal[i][j] - DeltaNet.CalVal[i][j]) * CalVal[i][j] * CalVal[i - 1][k] * StepK;
					}
				}
			}
		}
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i + 1]; j++)
			{
				b[i][j] += DeltaNet.b[i][j] / (CalFloat)n;
			}
		}
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i]; j++)
			{
				for (int k = 0; k < Size[i + 1]; k++)
				{
					w[i][j][k] += DeltaNet.w[i][j][k] / (CalFloat)n;
				}
			}
		}
		if (DebugTag)
		{
			DeltaNet.WriteToFile(FileName);
			FILE* fp;
			fp = fopen(FileName.c_str(), "a");
			for (int i = 0; i < Deep; i++)
			{
				for (int j = 0; j < Size[i + 1]; j++)
				{
					fprintf(fp, "%llf ", CalVal[i][j]);
				}
				fprintf(fp, "\n");
			}
			fclose(fp);
		}
		//free(&DeltaNet);
		//free(&VecSize);
	}
	void BPTrain(NeuroNet* DeltaNet, CalFloat** Input, int* Output, int OffsetIdx = 0, int n = 1, CalFloat StepK = 0.0010, bool DebugTag = false, string FileName = "BPDebug.txt")
	{
		for (int i = 0; i < DeltaNet->Deep; i++)
		{
			for (int j = 0; j < DeltaNet->Size[i + 1]; j++)
			{
				DeltaNet->b[i][j] = 0;
				DeltaNet->CalVal[i][j] = 0;
			}
		}
		for (int i = 0; i < DeltaNet->Deep; i++)
		{
			for (int j = 0; j < DeltaNet->Size[i]; j++)
			{
				for (int k = 0; k < DeltaNet->Size[i + 1]; k++)
				{
					DeltaNet->w[i][j][k] = 0;
				}
			}
		}

		for (int ImgIdx = 0; ImgIdx < n; ImgIdx++)
		{
			for (int i = 0; i < Deep; i++)
			{
				for (int j = 0; j < Size[i + 1]; j++)
				{
					CalVal[i][j] = 0;
				}
			}
			for (int i = 0; i < Deep; i++)
			{
				for (int j = 0; j < Size[i + 1]; j++)
				{
					CalFloat TmpVal = 0;
					for (int k = 0; k < Size[i]; k++)
					{
						if (i == 0)
						{
							TmpVal += Input[ImgIdx + OffsetIdx][k] * w[i][k][j];
						}
						else
						{
							TmpVal += CalVal[i - 1][k] * w[i][k][j];
						}
					}
					TmpVal += b[i][j];
					CalVal[i][j] = Sigmoid(TmpVal);
				}
			}
			CalFloat MaxVal = 0, MaxLabel = 0;//-1e9?
			for (int i = 0; i < Size[Deep]; i++)
			{
				if (MaxVal < CalVal[Deep - 1][i])
				{
					MaxLabel = i;
					MaxVal = CalVal[Deep - 1][i];
				}
			}
			for (int i = Deep - 1; i >= 0; i--)
			{
				if (i == Deep - 1)
				{
					for (int j = 0; j < Size[Deep]; j++)
					{
						DeltaNet->CalVal[Deep - 1][j] = (j == Output[ImgIdx + OffsetIdx] ? 1.00 : 0.00);//
					}
				}
				else
				{
					for (int k = 0; k < Size[i + 1]; k++)
					{
						CalFloat CalGrad = 0;
						for (int j = 0; j < Size[i + 1 + 1]; j++)
						{
							CalGrad += 2.00 * (CalVal[i + 1][j] - DeltaNet->CalVal[i + 1][j]) * CalVal[i + 1][j] * (1.00 - CalVal[i + 1][j]) * w[i + 1][k][j];
						}
						DeltaNet->CalVal[i][k] = CalVal[i][k] - StepK * CalGrad;//+= =
					}
				}//
				for (int j = 0; j < Size[i + 1]; j++)
				{
					DeltaNet->b[i][j] += -2.00 * (CalVal[i][j] - DeltaNet->CalVal[i][j]) * CalVal[i][j] * (1.00 - CalVal[i][j]) * StepK;
				}
				for (int k = 0; k < Size[i]; k++)
				{
					for (int j = 0; j < Size[i + 1]; j++)
					{
						if (i - 1 < 0) DeltaNet->w[i][k][j] += -2.00 * (CalVal[i][j] - DeltaNet->CalVal[i][j]) * CalVal[i][j] * (1.00 - CalVal[i][j]) * Input[ImgIdx + OffsetIdx][k] * StepK;
						else DeltaNet->w[i][k][j] += -2.00 * (CalVal[i][j] - DeltaNet->CalVal[i][j]) * CalVal[i][j] * (1.00 - CalVal[i][j]) * CalVal[i - 1][k] * StepK;
					}
				}
			}
		}
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i + 1]; j++)
			{
				b[i][j] += DeltaNet->b[i][j] / (CalFloat)n;
			}
		}
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i]; j++)
			{
				for (int k = 0; k < Size[i + 1]; k++)
				{
					w[i][j][k] += DeltaNet->w[i][j][k] / (CalFloat)n;
				}
			}
		}
		if (DebugTag)
		{
			DeltaNet->WriteToFile(FileName);
			FILE* fp;
			fp = fopen(FileName.c_str(), "a");
			for (int i = 0; i < Deep; i++)
			{
				for (int j = 0; j < Size[i + 1]; j++)
				{
					fprintf(fp, "%llf ", CalVal[i][j]);
				}
				fprintf(fp, "\n");
			}
			fclose(fp);
		}
		//free(&DeltaNet);
		//free(&VecSize);
	}
	void WriteToFile(string FileName)
	{
		FILE* fp;
		fp = fopen(FileName.c_str(), "w");
		fprintf(fp, "%d\n", Deep);
		for (int i = 0; i < Deep + 1; i++)
		{
			fprintf(fp, "%d ", Size[i]);
		}
		fprintf(fp, "\n");
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i + 1]; j++)
			{
				fprintf(fp, "%lf ", b[i][j]);
			}
			fprintf(fp, "\n");
		}
		fprintf(fp, "\n");
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i]; j++)
			{
				for (int k = 0; k < Size[i + 1]; k++)
				{
					fprintf(fp, "%lf ", w[i][j][k]);
				}
				fprintf(fp, "\n");
			}
			fprintf(fp, "\n");
		}
		fclose(fp);
	}
	void ReadFromFile(string FileName)
	{
		FILE* fp;
		fp = fopen(FileName.c_str(), "r");
		if (fp == NULL) {
			printf("FileOpenError\n");
			return;
		}
		//cout << "Reading from file:" << FileName << "\n";
		fscanf(fp, "%d", &Deep);
		for (int i = 0; i < Deep + 1; i++)
		{
			fscanf(fp, "%d", &Size[i]);
		}
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i + 1]; j++)
			{
				fscanf(fp, "%lf", &b[i][j]);//"%llf _"
				//cout << b[i][j] << " ";
			}
			//cout << "\n";
		}
		for (int i = 0; i < Deep; i++)
		{
			for (int j = 0; j < Size[i]; j++)
			{
				for (int k = 0; k < Size[i + 1]; k++)
				{
					fscanf(fp, "%lf", &w[i][j][k]);//
				}
			}
		}
		fclose(fp);
	}
};

unsigned int ImgFileTypeNumber, ImgCnt, ImgWidth, ImgHeight;
unsigned int TagFileTypeNumber, TagCnt;

unsigned int HighToLow(unsigned int Input)//大端转化成小端
{
	unsigned int Byte1, Byte2, Byte3, Byte4;
	Byte1 = Input & ((1 << 8) - 1);//(~(1 << 8));
	Input >>= 8;
	Byte2 = Input & ((1 << 8) - 1);
	Input >>= 8;
	Byte3 = Input & ((1 << 8) - 1);
	Input >>= 8;
	Byte4 = Input & ((1 << 8) - 1);
	Input >>= 8;
	return (Byte1 << 24) | (Byte2 << 16) | (Byte3 << 8) | (Byte4);
}

CalFloat** Img, ** TestImg;
int* Tag, * TestTag;

unsigned int TestImgFileTypeNumber, TestImgCnt, TestImgWidth, TestImgHeight;
unsigned int TestTagFileTypeNumber, TestTagCnt;

void ReadTestImg()
{
	FILE* fpTestImg, * fpTestTag;
	fpTestImg = fopen("D:\\data\\program\\NumberRecognizer\\NumberRecognizer\\NeuroNetwork\\DataBase\\t10k-images.idx3-ubyte", "rb");
	fread(&TestImgFileTypeNumber, sizeof(TestImgFileTypeNumber), 1, fpTestImg);
	fread(&TestImgCnt, sizeof(TestImgCnt), 1, fpTestImg);
	fread(&TestImgWidth, sizeof(TestImgWidth), 1, fpTestImg);
	fread(&TestImgHeight, sizeof(TestImgHeight), 1, fpTestImg);
	TestImgFileTypeNumber = HighToLow(TestImgFileTypeNumber);
	TestImgCnt = HighToLow(TestImgCnt);
	TestImgWidth = HighToLow(TestImgWidth);
	TestImgHeight = HighToLow(TestImgHeight);
	cout << TestImgFileTypeNumber << " " << TestImgCnt << " " << TestImgWidth << " " << TestImgHeight << "\n";

	fpTestTag = fopen("D:\\data\\program\\NumberRecognizer\\NumberRecognizer\\NeuroNetwork\\DataBase\\t10k-labels.idx1-ubyte", "rb");
	fread(&TestTagFileTypeNumber, sizeof(TestTagFileTypeNumber), 1, fpTestTag);
	fread(&TestTagCnt, sizeof(TestTagCnt), 1, fpTestTag);
	TestTagFileTypeNumber = HighToLow(TestTagFileTypeNumber);
	TestTagCnt = HighToLow(TestTagCnt);
	cout << TestTagFileTypeNumber << " " << TestTagCnt << "\n";

	TestImg = (CalFloat**)malloc(TestImgCnt * sizeof(CalFloat*));//free!
	for (int i = 0; i < TestImgCnt; i++)
	{
		TestImg[i] = (CalFloat*)malloc(TestImgWidth * TestImgHeight * sizeof(CalFloat));
	}
	TestTag = (int*)malloc(TestTagCnt * sizeof(int));

	Mat Image(TestImgHeight, TestImgWidth, CV_8UC3);

	for (int Idx = 0; Idx < TestImgCnt; Idx++)
	{
		if ((Idx + 1) % 10000 == 0) cout << "Reading:" << Idx + 1 << "\n";

		// 遍历图像的每个像素，并赋予不同的颜色值
		for (int y = 0; y < Image.rows; y++)
		{
			for (int x = 0; x < Image.cols; x++)
			{
				Vec3b Color;
				int PixelVal = 0;
				fread(&PixelVal, sizeof(char), 1, fpTestImg);
				TestImg[Idx][y * Image.rows + x] = PixelVal / 255.00;
				/*Color[0] = PixelVal; // 蓝色通道
				Color[1] = PixelVal;   // 绿色通道
				Color[2] = PixelVal;   // 红色通道

				// 将颜色值赋给图像的每个像素
				Image.at<Vec3b>(y, x) = Color;*/
			}
		}

		int CurTestTag = 0;
		fread(&CurTestTag, sizeof(char), 1, fpTestTag);
		TestTag[Idx] = CurTestTag;

		//putText(Image, to_string(CurTestTag), Point(0, 7), FONT_HERSHEY_TRIPLEX, 0.3, (255, 255, 255), 1);
		// 创建一个窗口显示图像
		/*namedWindow(to_string(CurTestTag), WINDOW_NORMAL);
		imshow(to_string(CurTestTag), Image);
		waitKey(0);
		destroyWindow(to_string(CurTestTag));*/
	}


	fclose(fpTestImg);
	fclose(fpTestTag);
}

void ReadImg()
{
	FILE* fpImg, * fpTag;
	fpImg = fopen("D:\\data\\program\\NumberRecognizer\\NumberRecognizer\\NeuroNetwork\\DataBase\\train-images.idx3-ubyte", "rb");
	fread(&ImgFileTypeNumber, sizeof(ImgFileTypeNumber), 1, fpImg);
	fread(&ImgCnt, sizeof(ImgCnt), 1, fpImg);
	fread(&ImgWidth, sizeof(ImgWidth), 1, fpImg);
	fread(&ImgHeight, sizeof(ImgHeight), 1, fpImg);
	ImgFileTypeNumber = HighToLow(ImgFileTypeNumber);
	ImgCnt = HighToLow(ImgCnt);
	ImgWidth = HighToLow(ImgWidth);
	ImgHeight = HighToLow(ImgHeight);
	cout << ImgFileTypeNumber << " " << ImgCnt << " " << ImgWidth << " " << ImgHeight << "\n";

	fpTag = fopen("D:\\data\\program\\NumberRecognizer\\NumberRecognizer\\NeuroNetwork\\DataBase\\train-labels.idx1-ubyte", "rb");
	fread(&TagFileTypeNumber, sizeof(TagFileTypeNumber), 1, fpTag);
	fread(&TagCnt, sizeof(TagCnt), 1, fpTag);
	TagFileTypeNumber = HighToLow(TagFileTypeNumber);
	TagCnt = HighToLow(TagCnt);
	cout << TagFileTypeNumber << " " << TagCnt << "\n";

	Img = (CalFloat**)malloc(ImgCnt * sizeof(CalFloat*));//free!
	for (int i = 0; i < ImgCnt; i++)
	{
		Img[i] = (CalFloat*)malloc(ImgWidth * ImgHeight * sizeof(CalFloat));
	}
	Tag = (int*)malloc(TagCnt * sizeof(int));

	Mat Image(ImgHeight, ImgWidth, CV_8UC3);

	for (int Idx = 0; Idx < ImgCnt; Idx++)
	{
		if ((Idx + 1) % 10000 == 0) cout << "Reading:" << Idx + 1 << "\n";

		// 遍历图像的每个像素，并赋予不同的颜色值
		for (int y = 0; y < Image.rows; y++)
		{
			for (int x = 0; x < Image.cols; x++)
			{
				Vec3b Color;
				int PixelVal = 0;
				fread(&PixelVal, sizeof(char), 1, fpImg);
				Img[Idx][y * Image.rows + x] = PixelVal / 255.00;
				/*Color[0] = PixelVal; // 蓝色通道
				Color[1] = PixelVal;   // 绿色通道
				Color[2] = PixelVal;   // 红色通道

				// 将颜色值赋给图像的每个像素
				Image.at<Vec3b>(y, x) = Color;*/
			}
		}

		int CurTag = 0;
		fread(&CurTag, sizeof(char), 1, fpTag);
		Tag[Idx] = CurTag;

		/*//putText(Image, to_string(CurTag), Point(0, 7), FONT_HERSHEY_TRIPLEX, 0.3, (255, 255, 255), 1);
		// 创建一个窗口显示图像
		namedWindow(to_string(CurTag), WINDOW_NORMAL);
		imshow(to_string(CurTag), Image);
		waitKey(0);
		destroyWindow(to_string(CurTag));*/
	}


	fclose(fpImg);
	fclose(fpTag);
}

#include <iostream>
#include <ctime>
#include <string>

string getCurrentDateTime()
{
	// 获取当前时间
	time_t currentTime = time(nullptr);
	tm* localTime = localtime(&currentTime);

	// 拼接年月日时分秒
	int year = localTime->tm_year + 1900;
	int month = localTime->tm_mon + 1;
	int day = localTime->tm_mday;
	int hour = localTime->tm_hour;
	int minute = localTime->tm_min;
	int second = localTime->tm_sec;

	// 构建时间字符串
	string dateTime = to_string(year)
		+ "_" + to_string(month)
		+ "_" + to_string(day)
		+ "_" + to_string(hour)
		+ "_" + to_string(minute)
		+ "_" + to_string(second);

	return dateTime;
}


int main()
{
	srand(time(NULL));
	vector<int> NeuroNetScale = { 784,16,16,10 };//
	//for (int i = 0; i < a.size(); i++) cout << a[i] << " ";
	NeuroNet NumRecogNet(NeuroNetScale, 1);//0
	//NumRecogNet.ReadFromFile(".\\Net_ACR_0.561200_Round_262600_2023_7_17_18_51_59.txt");//!!!
	NumRecogNet.WriteToFile("Test.txt");
	NeuroNet RWTestNumRecogNet(NeuroNetScale, 0);
	RWTestNumRecogNet.ReadFromFile(".\\Test.txt");
	RWTestNumRecogNet.WriteToFile("Test2.txt");
	ReadImg();
	ReadTestImg();
	int ACCount = 0;
	bool AllSame = true;
	int PredictFirstTag = 0;
	for (int i = 0; i < ImgCnt; i++)
	{
		if ((i + 1) % 5000 == 0)
		{
			cout << "Testing Sample:" << (i + 1) << "\n";
			/*for (int j = 0; j < 784; j++)
			{
				//cout << Img[i][j] << " ";
				if (Img[i][j] == 0) cout << "0 ";
				else cout << "1 ";
				if (j % 28 == 27) cout << "\n";
			}*/
		}
		int PredictTag = NumRecogNet.Run(Img[i], ((i + 1) % 5000 == 0 ? false : false));//
		if (i == 0)
		{
			PredictFirstTag = PredictTag;
		}
		if (PredictTag == Tag[i])
		{
			ACCount++;
		}
		if ((i + 1) % 5000 == 0) cout << "Result:" << PredictTag << " " << Tag[i] << "\n";
		if (PredictTag != PredictFirstTag)
		{
			AllSame = false;
		}
	}
	cout << "AC Rate:" << ACCount / (float)ImgCnt * 100 << "%\n";
	cout << "AllSame:" << AllSame << "\n";

	//Test Sigmoid
	/*for (int i = -100; i <= 100; i += 10)
	{
		cout << "Sigmoid(" << i << ")" << " == " << Sigmoid(i) << "    ";
		cout << "Sigmoid'(" << i << ")" << " == " << SigmoidGrad(i) << "\n";
	}*/

	const int TrainScale = 100000;
	//int TrainRound = TrainScale;
	long long TrainRound = 0;
	int GroupIdx = 0, int GroupNum = 50;
	//NumRecogNet.BPTrain(Img[GroupIdx], Tag + GroupIdx, GroupNum);
	//NumRecogNet.WriteToFile("Test3.txt");
	int BPCnt = 0;
	//int BPTest = 0;
	int BestACCount = 0;
	double LearningRate = 0.05;
	int ACBestRound = 0;
	int TestACCount = 0;
	while (true)//for(int Idx = 0;Idx < 5000;Idx++)
	{
		TrainRound++;
		if (TrainRound % 100 == 0) cout << "TrainRound:" << TrainRound << "  ";
		if (GroupIdx + GroupNum >= ImgCnt)
		{
			GroupIdx = 0;
		}

		//LearningRate = (double)(TrainRound - ACBestRound) / (double)30000.00;
		//LearningRate = LearningRate * LearningRate + 0.0001;
		//LearningRate = 0.00002 * LearningRate + 0.0001;
		double ACRate = TestACCount / (double)(TestImgCnt);
		LearningRate = 0.00002 * (double)(TrainRound - ACBestRound) * 1.00 / ((ACRate + 0.2) * (ACRate + 0.2)) + 0.00001;
		if (TrainRound - ACBestRound > 30000) LearningRate = 0.05;
		if (TrainRound - ACBestRound > 50000)
		{
			ACBestRound = TrainRound;
			BestACCount = TestACCount;
		}
		//0.2 -> 0.05
		NumRecogNet.BPTrain(&RWTestNumRecogNet, Img, Tag, GroupIdx, GroupNum, LearningRate, false);// (BPCnt == 9999 ? true : false), "DebugNet_" + getCurrentDateTime() + ".txt");
		BPCnt++;
		//BPTest++;
		if (BPCnt >= 100)
		{
			BPCnt = 0;
			TestACCount = 0;
			for (int i = 0; i < TestImgCnt; i++)
			{
				int PredictTag = NumRecogNet.Run(TestImg[i], ((i + 1) % 5000 == 0 ? false : false));//
				if (PredictTag == TestTag[i])
				{
					TestACCount++;
				}
			}
			if (((TestACCount - BestACCount) / (double)(TestImgCnt)) > 0.001)
			{
				BestACCount = TestACCount;
				NumRecogNet.WriteToFile("Net_ACR_" + to_string(TestACCount / (float)(TestImgCnt)) + "_Round_" + to_string(TrainRound) + "_" + getCurrentDateTime() + ".txt");
				ACBestRound = TrainRound;
			}
			cout << "ACRate=" << TestACCount / (float)(TestImgCnt) << "     BestACRate=" << BestACCount / (float)(TestImgCnt) << "   LearningRate=" << LearningRate << "\n";
		}
		/*if (BPTest >= 100)
		{
			BPTest = 0;
			int TestACCount = 0;
			for (int i = 0; i < TestImgCnt; i++)
			{
				int PredictTag = NumRecogNet.Run(TestImg[i], ((i + 1) % 5000 == 0 ? false : false));//
				if (PredictTag == TestTag[i])
				{
					TestACCount++;
				}
			}
			cout << "ACRate=" << TestACCount / (float)(TestImgCnt) << "\n";
		}*/
		//Sleep(100);
		GroupIdx++;//?
	}

	free(Img);
	free(Tag);
	return 0;
}