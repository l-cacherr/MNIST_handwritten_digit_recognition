﻿#include <iostream>
#include <string>
#include <vector>
#include <regex>
#include "dirent.h"
#include <utility>
#include "gnuplot-iostream.h"

using namespace gnuplotio;

bool compareFirstElement(const std::pair<long long, double>& pair1, const std::pair<long long, double>& pair2) {
    return pair1.first < pair2.first;
}

int main() {
    std::string directory = ".\\Files"; // 指定目录路径
    std::vector<std::string> txtFiles; // 用于存储txt文件名的向量

    DIR* dir;
    struct dirent* entry;

    if ((dir = opendir(directory.c_str())) != nullptr) {
        while ((entry = readdir(dir)) != nullptr) {
            std::string filename = entry->d_name;
            if (filename.length() >= 4 && filename.substr(filename.length() - 4) == ".txt") {
                txtFiles.push_back(filename);
            }
        }
        closedir(dir);
    }
    else {
        std::cout << "无法打开目录" << std::endl;
        return 1;
    }

    std::vector<std::string> arr = txtFiles;

    // 打印输出txt文件名
    /*for (const auto& file : txtFiles) {
        std::cout << file << std::endl;
    }*/

    std::vector<std::string> strArray = txtFiles;

    std::vector<std::string> filteredArray;
    std::vector<std::pair<long long, double>> extractedData;

    std::regex pattern("^Net_ACR_([0-9]+\\.[0-9]+)_Round_([0-9]+)_.*$");

    for (const std::string& str : strArray) {
        if (std::regex_match(str, pattern)) {
            filteredArray.push_back(str);

            std::smatch match;
            if (std::regex_search(str, match, pattern) && match.size() > 2) {
                double floatValue = std::stod(match[1].str());
                long long intValue = std::stoll(match[2].str());
                extractedData.push_back(std::make_pair(intValue, floatValue));
            }
        }
    }

    /*std::cout << "Filtered Array:" << std::endl;
    for (const std::string& str : filteredArray) {
        std::cout << str << std::endl;
    }

    std::cout << "Extracted Data:" << std::endl;
    for (const auto& data : extractedData) {
        std::cout << "Pair: (" << data.first << ", " << data.second << ")" << std::endl;
    }*/

    std::sort(extractedData.begin(), extractedData.end(), compareFirstElement);

    /*std::cout << "Extracted Data:" << std::endl;
    for (const auto& data : extractedData) {
        std::cout << "Pair: (" << data.first << ", " << data.second << ")" << std::endl;
    }*/

    std::vector<std::pair<double, double>> data(extractedData.size());

    for (int i = 0; i < data.size(); i++)
    {
        data[i].first = extractedData[i].first;
        data[i].second = extractedData[i].second;
        //std::cout << "Pair: (" << data[i].first << ", " << data[i].second << ")" << std::endl;
    }



    // 创建一个gnuplot对象
    Gnuplot gp;

    // 设置输出文件的格式
    gp << "set terminal png\n";

    // 设置输出文件的名称和大小
    gp << "set output 'line_plot.png'\n";
    gp << "set size ratio " << (std::to_string(data[data.size()].first / data[data.size()].second)) << "\n";
    //gp << "set size ratio -1\n";

    // 设置横坐标和纵坐标的范围
    gp << "set xrange [" << data.front().first << ":" << data.back().first << "]\n";
    gp << "set yrange [" << data.front().second << ":" << data.back().second << "]\n";
    
    // 绘制折线图
    gp << "plot '-' with linespoints title 'Data'\n";
    gp.send1d(data);
    return 0;
}
